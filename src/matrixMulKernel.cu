#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include "../include/utils.hpp"

using namespace std;

__global__ void MatrixMulKernel(float *M, float *N, float *P, int Width)
{
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    if ((Row < Width) && (Col < Width))
    {
        float Pvalue = 0;
        for (int k = 0; k < Width; ++k)
        {
            Pvalue += M[Row * Width + k] * N[k * Width + Col];
        }
        P[Row * Width + Col] = Pvalue;
    }
}

void launchKernel(int width, bool debug)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int size = width * width * sizeof(float);

    vector<float> M(size), N(size), P(size);

    generateRandomMatrix(M, width);
    generateRandomMatrix(N, width);

    float *h_M = M.data();
    float *h_N = N.data();
    float *h_P = P.data();

    float *d_M, *d_N, *d_P;

    hipMalloc((void **)&d_M, size);
    hipMalloc((void **)&d_N, size);
    hipMalloc((void **)&d_P, size);

    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);
    hipMemcpy(d_P, h_P, size, hipMemcpyHostToDevice);

    int blockDim = 16;
    int gridDim = ceil(width / blockDim) + 1;

    dim3 dimGrid(gridDim, gridDim, 1);
    dim3 dimBlock(blockDim, blockDim, 1);

    hipEventRecord(start);
    MatrixMulKernel<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, width);
    hipEventRecord(stop);

    hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);

    if (debug)
    {
        cout << "M: \n";
        printMatrix(M, width);
        cout << '\n';
        cout << "N: \n";
        printMatrix(N, width);
        cout << '\n';
        cout << "P: \n";
        printMatrix(P, width);
        cout << '\n';
    }
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);

    hipEventSynchronize(stop);

    float miliseconds = 0;
    hipEventElapsedTime(&miliseconds, start, stop);
    cout << "miliseconds: " << miliseconds << '\n';
}

int main(int argc, char *argv[])
{
    bool debug = false;
    if (argc < 2)
    {
        cout << "Usage: matrixMultKernel <n> \n";
        return 0;
    }
    else if (argc == 3)
    {
        if (string(argv[2]) == "-DEBUG")
        {
            debug = true;
        }
        else
        {
            cout << "Usage: matrixMultKernel <n> -DEBUG\n";
            return 0;
        }
    }

    int width = stoi(argv[1]);

    launchKernel(width, debug);
}